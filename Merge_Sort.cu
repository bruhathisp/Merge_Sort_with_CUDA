#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <thrust/sort.h>
#include <thrust/merge.h>

// Define constants
#define NUM_SETS 100000
#define DSIZE 100
typedef int mytype;

// Macro for ascending sorted data comparison
#define cmp(X,Y) ((X)<(Y))
#define THREADS_PER_BLOCK 512 // Threads per block
#define BLOCKS_PER_GRID 128 // Blocks per grid

#include <time.h>
#include <sys/time.h>
#define USECPSEC 1000000ULL // Microseconds per second

// Function to get current time in microseconds
long long time_usec(unsigned long long start){
  timeval tv;
  gettimeofday(&tv, 0);
  return ((tv.tv_sec*USECPSEC)+tv.tv_usec)-start;
}

// Merge function for sorted arrays on host and device
template <typename T>
__host__ __device__ void merge_arrays(const T * __restrict__  arr1, const T * __restrict__ arr2, T * __restrict__ result, const unsigned len_arr1, const unsigned len_arr2, const unsigned stride_arr1 = 1, const unsigned stride_arr2 = 1, const unsigned stride_result = 1){

  unsigned len_result = len_arr1 + len_arr2;
  unsigned index_result = 0;
  unsigned index_arr1 = 0;
  unsigned index_arr2 = 0;
  unsigned finished_arr1 = (len_arr2 == 0);
  unsigned finished_arr2 = (len_arr1 == 0);
  T next_arr1 = arr1[0];
  T next_arr2 = arr2[0];
  
  while (index_result < len_result){
    if (finished_arr1) {
      result[stride_result * index_result++] = next_arr1;
      index_arr1++;
      next_arr1 = arr1[stride_arr1 * index_arr1];
    }
    else if (finished_arr2) {
      result[stride_result * index_result++] = next_arr2;
      index_arr2++;
      next_arr2 = arr2[stride_arr2 * index_arr2];
    }
    else if (cmp(next_arr1, next_arr2)) {
      result[stride_result * index_result++] = next_arr1;
      index_arr1++;
      if (index_arr1 == len_arr1) finished_arr2++;
      else next_arr1 = arr1[stride_arr1 * index_arr1];
    }
    else {
      result[stride_result * index_result++] = next_arr2;
      index_arr2++;
      if (index_arr2 == len_arr2) finished_arr1++;
      else next_arr2 = arr2[stride_arr2 * index_arr2];
    }
  }
}

// Kernel function to perform row-major merge test on device
template <typename T>
__global__ void row_major_merge(const T * __restrict__  arr1, const T * __restrict__ arr2, T * __restrict__  result, int num_arrays, int array_length){

  int index = threadIdx.x + blockDim.x * blockIdx.x;

  while (index < num_arrays){
    int selected_index = index * array_length;
    merge_arrays(arr1 + selected_index, arr2 + selected_index, result + (2 * selected_index), array_length, array_length);
    index += blockDim.x * gridDim.x;
  }
}

// Kernel function to perform column-major merge test on device
template <typename T>
__global__ void column_major_merge(const T * __restrict__ arr1, const T * __restrict__ arr2, T * __restrict__ result, int num_arrays, int array_length, int stride_arr1, int stride_arr2, int stride_result){
  int index = threadIdx.x + blockDim.x * blockIdx.x;
  while (index < num_arrays){
    merge_arrays(arr1 + index, arr2 + index, result + index, array_length, array_length, stride_arr1, stride_arr2, stride_result);
    index += blockDim.x * gridDim.x;
  }
}

// Function to validate row-major merge results
template <typename T>
int validate_row_major(T *arr1, T *arr2, T *result, int num_arrays, int array_length){

  T *validation_array = (T *)malloc(2 * array_length * sizeof(T));
  for (int i = 0; i < num_arrays; i++){
    thrust::merge(arr1 + (i * array_length), arr1 + ((i + 1) * array_length), arr2 + (i * array_length), arr2 + ((i + 1) * array_length), validation_array);
#ifndef TIMING
    for (int j = 0; j < array_length * 2; j++)
      if (validation_array[j] != result[(i * 2 * array_length) + j]) {
        printf("row-major mismatch i: %d, j: %d, was: %d, should be: %d\n", i, j, result[(i * 2 * array_length) + j], validation_array[j]);
        return 0;
      }
#endif
  }
  return 1;
}

// Function to validate column-major merge results
template <typename T>
int validate_column_major(const T *result1, const T *result2, int num_arrays, int array_length){
  for (int i = 0; i < num_arrays; i++)
    for (int j = 0; j < 2 * array_length; j++)
      if (result1[i * (2 * array_length) + j] != result2[j * (num_arrays) + i]) {
        printf("column-major mismatch i: %d, j: %d, was: %d, should be: %d\n", i, j, result2[j * (num_arrays) + i], result1[i * (2 * array_length) + j]);
        return 0;
      }
  return 1;
}

// Main function
int main(){
  // Allocate host and device memory
  mytype *host_array1, *host_array2, *host_result, *device_array1, *device_array2, *device_result;
  host_array1 = (mytype *)malloc(DSIZE * NUM_SETS * sizeof(mytype));
  host_array2 = (mytype *)malloc(DSIZE * NUM_SETS * sizeof(mytype));
  host_result = (mytype *)malloc(DSIZE * NUM_SETS * sizeof(mytype) * 2);
  hipMalloc(&device_array1, (DSIZE * NUM_SETS + 1) * sizeof(mytype));
  hipMalloc(&device_array2, (DSIZE * NUM_SETS + 1) * sizeof(mytype));
  hipMalloc(&device_result, DSIZE * NUM_SETS * sizeof(mytype) * 2);

  // Test "row-major" storage
  for (int i = 0; i < DSIZE * NUM_SETS; i++){
    host_array1[i] = rand();
    host_array2[i] = rand();
  }
  thrust::sort(host_array1, host_array1 + DSIZE * NUM_SETS);
  thrust::sort(host_array2, host_array2 + DSIZE * NUM_SETS);
  hipMemcpy(device_array1, host_array1, DSIZE * NUM_SETS * sizeof(mytype), hipMemcpyHostToDevice);
  hipMemcpy(device_array2, host_array2, DSIZE * NUM_SETS * sizeof(mytype), hipMemcpyHostToDevice);
  unsigned long gpu_time = time_usec(0);
  row_major_merge<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK>>>(device_array1, device_array2, device_result, NUM_SETS, DSIZE);
  hipDeviceSynchronize();
  gpu_time = time_usec(gpu_time);
  hipMemcpy(host_result, device_result, DSIZE * NUM_SETS * 2 * sizeof(mytype), hipMemcpyDeviceToHost);
  unsigned long cpu_time = time_usec(0);
  if (!validate_row_major(host_array1, host_array2, host_result, NUM_SETS, DSIZE)) {
    printf("row-major validation failed!\n");
    return 1;
  }
  cpu_time = time_usec(cpu_time);
  printf("CPU time: %f, GPU row-major time: %f\n", cpu_time / (float)USECPSEC, gpu_time / (float)USECPSEC);

  // Test "column-major" storage
  mytype *host_array_col1, *host_array_col2, *host_result_col;
  host_array_col1 = (mytype *)malloc(DSIZE * NUM_SETS * sizeof(mytype));
  host_array_col2 = (mytype *)malloc(DSIZE * NUM_SETS * sizeof(mytype));
  host_result_col = (mytype *)malloc(DSIZE * NUM_SETS * sizeof(mytype));
  for (int i = 0; i < NUM_SETS; i++)
    for (int j = 0; j < DSIZE; j++){
      host_array_col1[j * NUM_SETS + i] = host_array1[i * DSIZE + j];
      host_array_col2[j * NUM_SETS + i] = host_array2[i * DSIZE + j];
    }
  hipMemcpy(device_array1, host_array_col1, DSIZE * NUM_SETS * sizeof(mytype), hipMemcpyHostToDevice);
  hipMemcpy(device_array2, host_array_col2, DSIZE * NUM_SETS * sizeof(mytype), hipMemcpyHostToDevice);
  gpu_time = time_usec(0);
  column_major_merge<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK>>>(device_array1, device_array2, device_result, NUM_SETS, DSIZE, NUM_SETS, NUM_SETS, NUM_SETS);
  hipDeviceSynchronize();
  gpu_time = time_usec(gpu_time);
  hipMemcpy(host_result_col, device_result, DSIZE * NUM_SETS * 2 * sizeof(mytype), hipMemcpyDeviceToHost);
  if (!validate_column_major(host_result, host_result_col, NUM_SETS, DSIZE)) {
    printf("column-major validation failed!\n");
    return 1;
  }

  printf("GPU column-major time: %f\n", gpu_time / (float)USECPSEC);
  return 0;
}
